#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <stdio.h>
#include "MWCNNLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t VCbcPxtPsBLTrHYdEvqn; void 
curand_call_line_file(hiprandStatus_t qWwjVYwfnvEnFKlgpqwA, const int 
eFaDPmxDdzHlRYSAoMmX, const char *UEESbUvbMihFnquvuFij) { if (qWwjVYwfnvEnFKlgpqwA != 
HIPRAND_STATUS_SUCCESS) { printf("%d, line: %d, file: %s\n", qWwjVYwfnvEnFKlgpqwA, 
eFaDPmxDdzHlRYSAoMmX, UEESbUvbMihFnquvuFij); exit(EXIT_FAILURE); } }
#endif
 float* malloc_call_line_file(size_t msize, const int eFaDPmxDdzHlRYSAoMmX, const 
char *UEESbUvbMihFnquvuFij) { float * mem = (float*)malloc(msize); if (!mem) { 
printf("%s, line: %d, file: %s\n", "Memory allocation failed. ", 
eFaDPmxDdzHlRYSAoMmX, UEESbUvbMihFnquvuFij); exit(EXIT_FAILURE); } return mem; } void 
call_cuda_free(float* mem) { hipError_t qWwjVYwfnvEnFKlgpqwA = hipFree(mem); if 
(qWwjVYwfnvEnFKlgpqwA != hipErrorDeinitialized) { CUDA_CALL(qWwjVYwfnvEnFKlgpqwA); 
} } void cuda_call_line_file(hipError_t qWwjVYwfnvEnFKlgpqwA, const int 
eFaDPmxDdzHlRYSAoMmX, const char *UEESbUvbMihFnquvuFij) { if (qWwjVYwfnvEnFKlgpqwA != 
hipSuccess) { printf("%s, line: %d, file: %s\n", 
hipGetErrorString(qWwjVYwfnvEnFKlgpqwA), eFaDPmxDdzHlRYSAoMmX, UEESbUvbMihFnquvuFij); 
exit(EXIT_FAILURE); } } void cudnn_call_line_file(hipdnnStatus_t 
qWwjVYwfnvEnFKlgpqwA, const int eFaDPmxDdzHlRYSAoMmX, const char *UEESbUvbMihFnquvuFij) { if 
(qWwjVYwfnvEnFKlgpqwA != HIPDNN_STATUS_SUCCESS) { 
printf("%s, line: %d, file: %s\n", hipdnnGetErrorString(qWwjVYwfnvEnFKlgpqwA), 
eFaDPmxDdzHlRYSAoMmX, UEESbUvbMihFnquvuFij); exit(EXIT_FAILURE); } } const char* 
cublasGetErrorString(hipblasStatus_t qWwjVYwfnvEnFKlgpqwA) { 
switch(qWwjVYwfnvEnFKlgpqwA) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE";  case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH";  case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED";  case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR";  case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED";  case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN";  } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t qWwjVYwfnvEnFKlgpqwA, const int 
eFaDPmxDdzHlRYSAoMmX, const char *UEESbUvbMihFnquvuFij) { if (qWwjVYwfnvEnFKlgpqwA != 
HIPBLAS_STATUS_SUCCESS) { printf("%s, line: %d, file: %s\n", 
cublasGetErrorString(qWwjVYwfnvEnFKlgpqwA), eFaDPmxDdzHlRYSAoMmX, UEESbUvbMihFnquvuFij); 
exit(EXIT_FAILURE); } } MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl) : SUleyRyvAggTFnSdxLru(0.0), SGsAudmgjmvcUXzzrUtf(1.0), 
SDWKEQTZaTFZByPlzUDR(-1.0), dMxIKDGTITyhdLqIHBLA(layer), 
fYaOQTeunPwVjnhhTECh(ntwk_impl), RAtlBpdedvgxUsgDTsch(0)  { } 
MWCNNLayerImpl::~MWCNNLayerImpl() { for(std::map<int, 
hipdnnTensorDescriptor_t*>::iterator it = lteHjcLsItGbVPMQtGDB.begin(); it != 
lteHjcLsItGbVPMQtGDB.end(); ++it) { delete it->second; it->second = 0; } } 
float* MWCNNLayerImpl::getZeroPtr() { return &SUleyRyvAggTFnSdxLru; } float* 
MWCNNLayerImpl::getOnePtr() { return &SGsAudmgjmvcUXzzrUtf; } float* 
MWCNNLayerImpl::getNegOnePtr() { return &SDWKEQTZaTFZByPlzUDR; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getOutputDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lteHjcLsItGbVPMQtGDB.find(index); if (it == lteHjcLsItGbVPMQtGDB.end()) { 
hipdnnTensorDescriptor_t* tmp = new hipdnnTensorDescriptor_t;  
lteHjcLsItGbVPMQtGDB[index] = tmp; return tmp; } else { return it->second; } } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getCuDNNDescriptor(MWTensor* tensor) { 
MWCNNLayerImpl* impl = tensor->getOwner()->getImpl(); if (!impl || 
dynamic_cast<MWPassthroughLayer*>(tensor->getOwner())) { 
assert(dynamic_cast<MWPassthroughLayer*>(tensor->getOwner())); return 
getCuDNNDescriptor(tensor->getOwner()->getInputTensor(0)); } return 
impl->getOutputDescriptor(tensor->getSourcePortIndex()); } void __global__ 
__launch_bounds__(1024) padInputImpl(float* in, int inputH, int inputW, int 
inputCh, int outputH, int outputW, int offsetH, int offsetW, float* out, int 
inputElems) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
inputElems; i+= blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); 
int rem = (i - idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); 
int rem1 = rem - idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = 
rem1 - idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = 
idxH + offsetH; int outputCol = idxCol + offsetW; int outputCh = inputCh; *(out 
+ idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + outputR*(outputW) 
+ outputCol) = *(in + i); } } } void MWCNNLayerImpl::padInput(float* 
XLJXOFXdnZOyJvtltbyr, int bDTIjtxZiSHtjwzgEluE, int bMAyVFGSPDjmUbziYLAy, int 
atVCyzqXZAZxwlkRLBRA, int nNULvWnBXnnWdpEkHPAH, int nlIRrOJaFuVaywxOqOyb, int 
jaqKGCwoANNDMHgAsehk, int jhFUWlztBndwjbXwYNaJ, float* kNsviQGMPdXzNMRixGWR, int 
gzSTokDHvkXefhiGDcWL) { int tqZLvfMHdgZzbchUyDzd = (gzSTokDHvkXefhiGDcWL < 
1024) ? gzSTokDHvkXefhiGDcWL : 1024; int NldNILHvuQqQPSAHXxdT = 
(gzSTokDHvkXefhiGDcWL + tqZLvfMHdgZzbchUyDzd - 
1)/tqZLvfMHdgZzbchUyDzd; padInputImpl<<<NldNILHvuQqQPSAHXxdT, 
tqZLvfMHdgZzbchUyDzd>>>(XLJXOFXdnZOyJvtltbyr, bDTIjtxZiSHtjwzgEluE, 
bMAyVFGSPDjmUbziYLAy, atVCyzqXZAZxwlkRLBRA, nNULvWnBXnnWdpEkHPAH, nlIRrOJaFuVaywxOqOyb, 
jaqKGCwoANNDMHgAsehk, jhFUWlztBndwjbXwYNaJ, kNsviQGMPdXzNMRixGWR, gzSTokDHvkXefhiGDcWL); } 
MWInputLayerImpl::MWInputLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int fSKMHAqIghbYYgyIpNDw, int WprSrhAStKGxyXeoxETy, int vjDFlBZzKvbpPseAtMBP, int 
OumvfgWXDdmsQaciHMHx, int xHViLEwTujGGrPZZgmbF, const char* avg_file_name, int outbufIdx) 
: MWCNNLayerImpl(layer, ntwk_impl) { createInputLayer(fSKMHAqIghbYYgyIpNDw, 
WprSrhAStKGxyXeoxETy, vjDFlBZzKvbpPseAtMBP, OumvfgWXDdmsQaciHMHx, xHViLEwTujGGrPZZgmbF, avg_file_name, 
outbufIdx); } MWInputLayerImpl::~MWInputLayerImpl() { } void 
MWInputLayerImpl::createInputLayer(int fSKMHAqIghbYYgyIpNDw, int WprSrhAStKGxyXeoxETy, int 
vjDFlBZzKvbpPseAtMBP, int OumvfgWXDdmsQaciHMHx, int xHViLEwTujGGrPZZgmbF, const char* 
avg_file_name, int outbufIdx){ if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*WprSrhAStKGxyXeoxETy*vjDFlBZzKvbpPseAtMBP*OumvfgWXDdmsQaciHMHx*fSKMHAqIghbYYgyIpNDw)); } else { 
setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
getLayer()->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&MIBnYCbKBdUrlfqlHdoo)); 
dJcdBfQQLhIAYHPxwQeg = xHViLEwTujGGrPZZgmbF; 
fYaOQTeunPwVjnhhTECh->setWorkSpaceSize(0); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fSKMHAqIghbYYgyIpNDw, OumvfgWXDdmsQaciHMHx, WprSrhAStKGxyXeoxETy, 
vjDFlBZzKvbpPseAtMBP)); if( dJcdBfQQLhIAYHPxwQeg == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(MIBnYCbKBdUrlfqlHdoo, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, OumvfgWXDdmsQaciHMHx, WprSrhAStKGxyXeoxETy, vjDFlBZzKvbpPseAtMBP)); 
CUDA_CALL(hipMalloc((void**)&IbSWJNMuIiKbocfQKqXb, 
sizeof(float)*OumvfgWXDdmsQaciHMHx*WprSrhAStKGxyXeoxETy*vjDFlBZzKvbpPseAtMBP)); int fhikqqlnUKCjleVKDqiG = 
OumvfgWXDdmsQaciHMHx*WprSrhAStKGxyXeoxETy*vjDFlBZzKvbpPseAtMBP;  loadAvg(avg_file_name, 
fhikqqlnUKCjleVKDqiG); } else if (dJcdBfQQLhIAYHPxwQeg == 2){ 
CUDA_CALL(hipMalloc((void**)&IbSWJNMuIiKbocfQKqXb, sizeof(float)*OumvfgWXDdmsQaciHMHx)); int 
fhikqqlnUKCjleVKDqiG = OumvfgWXDdmsQaciHMHx;  loadAvg(avg_file_name, fhikqqlnUKCjleVKDqiG); }
#ifdef RANDOM
 hiprandGenerateUniform(VCbcPxtPsBLTrHYdEvqn, MW_data, fSKMHAqIghbYYgyIpNDw*OumvfgWXDdmsQaciHMHx*WprSrhAStKGxyXeoxETy*vjDFlBZzKvbpPseAtMBP);
#endif
 fYaOQTeunPwVjnhhTECh->setWorkSpaceSize(0); return; } void 
MWInputLayerImpl::loadAvg(const char* UKtMXCCqdjeyaVHabkxg, int fhikqqlnUKCjleVKDqiG) 
{ FILE* UzaGmBLFEwmwaFXebUma = MWCNNLayer::openBinaryFile(UKtMXCCqdjeyaVHabkxg); 
assert(UzaGmBLFEwmwaFXebUma); float* OKaRVOctKLlnIyGmjRNW = 
MALLOC_CALL(sizeof(float)*fhikqqlnUKCjleVKDqiG); fread(OKaRVOctKLlnIyGmjRNW, 
sizeof(float), fhikqqlnUKCjleVKDqiG, UzaGmBLFEwmwaFXebUma); 
CUDA_CALL(hipMemcpy(IbSWJNMuIiKbocfQKqXb, OKaRVOctKLlnIyGmjRNW, 
sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice)); 
free(OKaRVOctKLlnIyGmjRNW); fclose(UzaGmBLFEwmwaFXebUma); return; } void __global__ 
__launch_bounds__(1024) subtractMeanPerChannelImpl(float * 
eybNKlJCSDUvsznWynwK, float * REXdEoRjxuQJkqgIDihy, const int 
fxxCPKTclxXPxrdMAkwi, const int fvTCtkwXgyScJYogJVFU, const int 
CDJtexcMbXMWAmnNZsNf) {  for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
CDJtexcMbXMWAmnNZsNf; i+= blockDim.x*gridDim.x) {  int idx = static_cast<int>((i % 
fvTCtkwXgyScJYogJVFU) / fxxCPKTclxXPxrdMAkwi); 
REXdEoRjxuQJkqgIDihy[i] -= eybNKlJCSDUvsznWynwK[idx]; } } void 
MWInputLayerImpl::predict() { if ( dJcdBfQQLhIAYHPxwQeg == 1) 
CUDNN_CALL(hipdnnAddTensor(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
getNegOnePtr(), MIBnYCbKBdUrlfqlHdoo, IbSWJNMuIiKbocfQKqXb, getOnePtr(), 
*getOutputDescriptor(), getData())); else if( dJcdBfQQLhIAYHPxwQeg == 2){ 
MWInputLayer* thisLayer = static_cast<MWInputLayer*>(getLayer()); MWTensor* 
opTensor = thisLayer->getOutputTensor(0); int fjfzkUfcCOqjrkAVGfuc = 
opTensor->getHeight()*opTensor->getWidth()* 
opTensor->getChannels()*opTensor->getBatchSize(); int 
fxxCPKTclxXPxrdMAkwi = opTensor->getHeight() * opTensor->getWidth(); 
int fvTCtkwXgyScJYogJVFU = 
fxxCPKTclxXPxrdMAkwi*opTensor->getChannels(); int 
tqZLvfMHdgZzbchUyDzd = (fjfzkUfcCOqjrkAVGfuc < 1024) ? fjfzkUfcCOqjrkAVGfuc : 
1024; int NldNILHvuQqQPSAHXxdT = (fjfzkUfcCOqjrkAVGfuc + 
tqZLvfMHdgZzbchUyDzd - 1)/tqZLvfMHdgZzbchUyDzd; 
subtractMeanPerChannelImpl<<<NldNILHvuQqQPSAHXxdT, 
tqZLvfMHdgZzbchUyDzd>>>( IbSWJNMuIiKbocfQKqXb, getData(), 
fxxCPKTclxXPxrdMAkwi, fvTCtkwXgyScJYogJVFU, fjfzkUfcCOqjrkAVGfuc); 
} return; } void MWInputLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < dMxIKDGTITyhdLqIHBLA->getNumOutputs(); idx++) {  float* data = 
dMxIKDGTITyhdLqIHBLA->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) call_cuda_free(data); 
} } if ( dJcdBfQQLhIAYHPxwQeg == 1) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(MIBnYCbKBdUrlfqlHdoo)); if (IbSWJNMuIiKbocfQKqXb) 
{ call_cuda_free(IbSWJNMuIiKbocfQKqXb); } } return; } 
MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int inPlace, int outbufIdx)  : MWCNNLayerImpl(layer, ntwk_impl) , 
XYbzSmRQGatVJtGmDZSo(inPlace)  { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&olKGEIcsxmLSoMhRhEtP)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createReLULayer(outbufIdx); } MWReLULayerImpl::~MWReLULayerImpl() { } void 
MWReLULayerImpl::createReLULayer(int outbufIdx) { MWReLULayer* reluLayer = 
static_cast<MWReLULayer*>(getLayer()); MWTensor* ipTensor = 
reluLayer->getInputTensor(0); MWTensor* opTensor = 
reluLayer->getOutputTensor(0); 
CUDNN_CALL(hipdnnSetActivationDescriptor(olKGEIcsxmLSoMhRhEtP, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));  
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); if 
(outbufIdx < 0) { if(XYbzSmRQGatVJtGmDZSo){ RAtlBpdedvgxUsgDTsch = 
getLayer()->getInputTensor()->getData(); } else{ 
CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*opTensor->getHeight()* 
opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); } } 
else { setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
reluLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); }  } void 
MWReLULayerImpl::predict() { MWReLULayer* reluLayer = 
static_cast<MWReLULayer*>(getLayer()); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(reluLayer->getInputTensor()); 
CUDNN_CALL(hipdnnActivationForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
olKGEIcsxmLSoMhRhEtP, getOnePtr(), ipDesc, 
reluLayer->getInputTensor()->getData(), getZeroPtr(), *getOutputDescriptor(), 
RAtlBpdedvgxUsgDTsch)); } void MWReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(olKGEIcsxmLSoMhRhEtP)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); MWTensor* op 
= getLayer()->getOutputTensor(0); float* data = op->getData(); if (data) { 
if((op->getopBufIndex() < 0) && !XYbzSmRQGatVJtGmDZSo) call_cuda_free(data); } } 
MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, unsigned IAlDgIFcchbwRGBSfVfA,  double AFQBkxwYGKLsACiDKwRM,  
double AHqhysOOIgbDpWZoPUFT,  double BUOdotSvmFyUWQKMUdra, int outbufIdx) : 
MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateLRNDescriptor(&fSbUUBgjKRbNXrHrlOLo)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createNormLayer(IAlDgIFcchbwRGBSfVfA, AFQBkxwYGKLsACiDKwRM, 
AHqhysOOIgbDpWZoPUFT, BUOdotSvmFyUWQKMUdra, outbufIdx); } 
MWNormLayerImpl::~MWNormLayerImpl() { } void MWNormLayerImpl::createNormLayer( 
unsigned IAlDgIFcchbwRGBSfVfA,  double AFQBkxwYGKLsACiDKwRM,  double 
AHqhysOOIgbDpWZoPUFT,  double BUOdotSvmFyUWQKMUdra, int outbufIdx) { MWNormLayer* normLayer 
= static_cast<MWNormLayer*>(getLayer()); MWTensor* ipTensor = 
normLayer->getInputTensor(0); MWTensor* opTensor = 
normLayer->getOutputTensor(0); int numOutputFeatures = opTensor->getChannels(); 
if (outbufIdx < 0) { CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*opTensor->getHeight()*opTensor->getWidth()*numOutputFeatures*opTensor->getBatchSize())); 
} else { setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
normLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnSetLRNDescriptor(fSbUUBgjKRbNXrHrlOLo, 
IAlDgIFcchbwRGBSfVfA, AFQBkxwYGKLsACiDKwRM, AHqhysOOIgbDpWZoPUFT, 
BUOdotSvmFyUWQKMUdra)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); return; 
} void MWNormLayerImpl::predict() { MWNormLayer* normLayer = 
static_cast<MWNormLayer*>(getLayer()); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(normLayer->getInputTensor()); 
CUDNN_CALL(hipdnnLRNCrossChannelForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
fSbUUBgjKRbNXrHrlOLo, HIPDNN_LRN_CROSS_CHANNEL, getOnePtr(), ipDesc, 
normLayer->getInputTensor()->getData(),getZeroPtr(), *getOutputDescriptor(), 
normLayer->getOutputTensor()->getData())); } void MWNormLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyLRNDescriptor(fSbUUBgjKRbNXrHrlOLo)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  MWTensor* op = 
getLayer()->getOutputTensor(idx); float* data = op->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0 ) 
call_cuda_free(data); } }  } void __global__ MWSetDyForBackPropImpl(float * 
SIBpKtDURUWQaaenbwrC, const int hljcfGWsvZXJZNrImpJB); void __global__ 
doMWMaxPoolingLayerImpl(float * ZinudJuZuGitiNTsJpBR, float * 
ZDWLzHUkuZuIUZHfbGDY, const int CDJtexcMbXMWAmnNZsNf); 
MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int GFienSVKLlDQuZeqAdLC,  int GeeOVBfQrpMacIFBLKOo,  
int GsZlHFuhbvjLtRMDjXnW,  int HJHXkKmgFxxIOsIvRRnF, int EvebzoroiuKkIxwjkGnD, int 
ECTnqgWHyHCHCLBZlffd,  int FrpxvsDMwwgbpqHXWxmN, int FwLnexHgxHRquTKmNpoa, 
bool JgLfgHrHMEMmMYTettJF, int iPqBiFnIJMxelVhQBZex, const std::vector<int>& 
NtWaRGCHLeTapjWdEHHS) : MWCNNLayerImpl(layer, ntwk_impl) , 
BRSPqxNffoBYKqpSVHne(JgLfgHrHMEMmMYTettJF) , ZinudJuZuGitiNTsJpBR(0) 
, SIBpKtDURUWQaaenbwrC(0) , ZDWLzHUkuZuIUZHfbGDY(0)  {  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&npGnQZLrEfVTQnEbwqij)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createMaxPoolingLayer(GFienSVKLlDQuZeqAdLC,GeeOVBfQrpMacIFBLKOo,GsZlHFuhbvjLtRMDjXnW,HJHXkKmgFxxIOsIvRRnF,EvebzoroiuKkIxwjkGnD,ECTnqgWHyHCHCLBZlffd,FrpxvsDMwwgbpqHXWxmN,FwLnexHgxHRquTKmNpoa, 
iPqBiFnIJMxelVhQBZex, NtWaRGCHLeTapjWdEHHS); } 
MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::createMaxPoolingLayer(int GFienSVKLlDQuZeqAdLC,  int 
GeeOVBfQrpMacIFBLKOo,  int GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, int 
EvebzoroiuKkIxwjkGnD, int ECTnqgWHyHCHCLBZlffd,  int FrpxvsDMwwgbpqHXWxmN, 
int FwLnexHgxHRquTKmNpoa, int iPqBiFnIJMxelVhQBZex, const std::vector<int>& 
NtWaRGCHLeTapjWdEHHS) { MWMaxPoolingLayer* maxpoolLayer = 
static_cast<MWMaxPoolingLayer*>(getLayer()); MWTensor* ipTensor = 
maxpoolLayer->getInputTensor(0); int oJUVMnJggjhEdQLWzIUC = 
EvebzoroiuKkIxwjkGnD; int oYbqYsqgVhrUzFEKbBbR = 
FrpxvsDMwwgbpqHXWxmN; hipdnnTensorDescriptor_t bUVPfnrJhLfHzOLUUrKk = 
*getCuDNNDescriptor(ipTensor);  
CUDNN_CALL(hipdnnSetPooling2dDescriptor(npGnQZLrEfVTQnEbwqij, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, GFienSVKLlDQuZeqAdLC, GeeOVBfQrpMacIFBLKOo, 
oJUVMnJggjhEdQLWzIUC, oYbqYsqgVhrUzFEKbBbR, GsZlHFuhbvjLtRMDjXnW, 
HJHXkKmgFxxIOsIvRRnF)); int fSKMHAqIghbYYgyIpNDw, OumvfgWXDdmsQaciHMHx, WprSrhAStKGxyXeoxETy, 
vjDFlBZzKvbpPseAtMBP; CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(npGnQZLrEfVTQnEbwqij, 
bUVPfnrJhLfHzOLUUrKk, &fSKMHAqIghbYYgyIpNDw ,&OumvfgWXDdmsQaciHMHx, &WprSrhAStKGxyXeoxETy, 
&vjDFlBZzKvbpPseAtMBP)); WprSrhAStKGxyXeoxETy = getLayer()->getOutputTensor(0)->getHeight(); 
vjDFlBZzKvbpPseAtMBP = getLayer()->getOutputTensor(0)->getWidth(); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fSKMHAqIghbYYgyIpNDw, OumvfgWXDdmsQaciHMHx, WprSrhAStKGxyXeoxETy, 
vjDFlBZzKvbpPseAtMBP)); int outbufIdx = NtWaRGCHLeTapjWdEHHS[0]; if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*fSKMHAqIghbYYgyIpNDw*OumvfgWXDdmsQaciHMHx*WprSrhAStKGxyXeoxETy*vjDFlBZzKvbpPseAtMBP)); } else { 
setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
maxpoolLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } if 
(BRSPqxNffoBYKqpSVHne){ 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(1))); const int 
fjfzkUfcCOqjrkAVGfuc = 
(ipTensor->getHeight())*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
CUDA_CALL(hipMalloc((void**)&ZinudJuZuGitiNTsJpBR, 
sizeof(float)*fjfzkUfcCOqjrkAVGfuc)); assert(iPqBiFnIJMxelVhQBZex == 2); int 
bufIndex = NtWaRGCHLeTapjWdEHHS[1]; if (bufIndex < 0) { 
CUDA_CALL(hipMalloc((void**)&ZDWLzHUkuZuIUZHfbGDY, 
sizeof(float)*fSKMHAqIghbYYgyIpNDw*OumvfgWXDdmsQaciHMHx*WprSrhAStKGxyXeoxETy*vjDFlBZzKvbpPseAtMBP)); } else { 
ZDWLzHUkuZuIUZHfbGDY = fYaOQTeunPwVjnhhTECh->memBuffer[bufIndex]; 
maxpoolLayer->getOutputTensor(1)->setopBufIndex(bufIndex); } 
assert((OumvfgWXDdmsQaciHMHx == ipTensor->getChannels()) && (fSKMHAqIghbYYgyIpNDw == 
ipTensor->getBatchSize()));  const int hljcfGWsvZXJZNrImpJB = 
vjDFlBZzKvbpPseAtMBP*WprSrhAStKGxyXeoxETy*OumvfgWXDdmsQaciHMHx*fSKMHAqIghbYYgyIpNDw; 
CUDA_CALL(hipMalloc((void**)&SIBpKtDURUWQaaenbwrC, 
sizeof(float)*hljcfGWsvZXJZNrImpJB)); int tqZLvfMHdgZzbchUyDzd = 
(hljcfGWsvZXJZNrImpJB < 1024) ? hljcfGWsvZXJZNrImpJB : 1024; int 
NldNILHvuQqQPSAHXxdT = (hljcfGWsvZXJZNrImpJB + tqZLvfMHdgZzbchUyDzd - 
1)/tqZLvfMHdgZzbchUyDzd; 
MWSetDyForBackPropImpl<<<NldNILHvuQqQPSAHXxdT, 
tqZLvfMHdgZzbchUyDzd>>>( SIBpKtDURUWQaaenbwrC, hljcfGWsvZXJZNrImpJB); } } void 
MWMaxPoolingLayerImpl::predict() { MWMaxPoolingLayer* maxpoolLayer = 
static_cast<MWMaxPoolingLayer*>(getLayer()); hipdnnTensorDescriptor_t 
bUVPfnrJhLfHzOLUUrKk = *getCuDNNDescriptor(maxpoolLayer->getInputTensor()); 
MWTensor* ipTensor = getLayer()->getInputTensor(0); 
CUDNN_CALL(hipdnnPoolingForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
npGnQZLrEfVTQnEbwqij, getOnePtr(), bUVPfnrJhLfHzOLUUrKk, ipTensor->getData(), 
getZeroPtr(), *getOutputDescriptor(), 
maxpoolLayer->getOutputTensor()->getData())); if (BRSPqxNffoBYKqpSVHne) { 
CUDNN_CALL(hipdnnPoolingBackward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
npGnQZLrEfVTQnEbwqij, getOnePtr(), *getOutputDescriptor(0), 
getLayer()->getOutputTensor(0)->getData(), *getOutputDescriptor(0), 
SIBpKtDURUWQaaenbwrC, bUVPfnrJhLfHzOLUUrKk, ipTensor->getData(), getZeroPtr(), 
bUVPfnrJhLfHzOLUUrKk, ZinudJuZuGitiNTsJpBR)); int fjfzkUfcCOqjrkAVGfuc = 
ipTensor->getHeight()*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
int tqZLvfMHdgZzbchUyDzd = (fjfzkUfcCOqjrkAVGfuc < 1024) ? 
fjfzkUfcCOqjrkAVGfuc : 1024; int NldNILHvuQqQPSAHXxdT = (fjfzkUfcCOqjrkAVGfuc + 
tqZLvfMHdgZzbchUyDzd - 1)/tqZLvfMHdgZzbchUyDzd; 
doMWMaxPoolingLayerImpl<<<NldNILHvuQqQPSAHXxdT, 
tqZLvfMHdgZzbchUyDzd>>>( ZinudJuZuGitiNTsJpBR, 
maxpoolLayer->getOutputTensor(1)->getData(), fjfzkUfcCOqjrkAVGfuc); } return; } 
void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(npGnQZLrEfVTQnEbwqij)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(BRSPqxNffoBYKqpSVHne){ 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(1))); } for(int 
idx = 0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) call_cuda_free(data); 
} } if (ZinudJuZuGitiNTsJpBR){ 
call_cuda_free(ZinudJuZuGitiNTsJpBR); } if (SIBpKtDURUWQaaenbwrC){ 
call_cuda_free(SIBpKtDURUWQaaenbwrC); }  } float* 
MWMaxPoolingLayerImpl::getIndexData()  { return ZDWLzHUkuZuIUZHfbGDY; } void 
__global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float * 
SIBpKtDURUWQaaenbwrC, const int hljcfGWsvZXJZNrImpJB) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < hljcfGWsvZXJZNrImpJB; i+= blockDim.x*gridDim.x) { 
SIBpKtDURUWQaaenbwrC[i] = i+1; } } void __global__ __launch_bounds__(1024) 
doMWMaxPoolingLayerImpl(float * ZinudJuZuGitiNTsJpBR, float * 
ZDWLzHUkuZuIUZHfbGDY, const int CDJtexcMbXMWAmnNZsNf) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < CDJtexcMbXMWAmnNZsNf; i+= blockDim.x*gridDim.x) { if 
(static_cast<int>(ZinudJuZuGitiNTsJpBR[i]) != 0){ 
ZDWLzHUkuZuIUZHfbGDY[static_cast<int>(ZinudJuZuGitiNTsJpBR[i])-1] = 
i; } } } MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int CpMjJjtGOeWOzwxpAAQP, const char* 
wMySyzzledUmSLTWhuYH,  const char* NZjOkZPwLzQsdEVkwMcX, int outbufIdx) : 
MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&NMMfJylfQjiIUAKhXCJb)); 
createFCLayer(CpMjJjtGOeWOzwxpAAQP, wMySyzzledUmSLTWhuYH, 
NZjOkZPwLzQsdEVkwMcX, outbufIdx); } MWFCLayerImpl::~MWFCLayerImpl() { } void 
MWFCLayerImpl::createFCLayer( int CpMjJjtGOeWOzwxpAAQP, const char* 
wMySyzzledUmSLTWhuYH, const char* NZjOkZPwLzQsdEVkwMcX, int outbufIdx) { 
MWFCLayer* fcLayer = static_cast<MWFCLayer*>(getLayer()); MWTensor* opTensor = 
fcLayer->getOutputTensor(0); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*fcLayer->getOutputTensor()->getBatchSize()*fcLayer->getOutputTensor()->getChannels())); 
} else { setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
fcLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDA_CALL(hipMalloc((void**)&vjDFlBZzKvbpPseAtMBP, 
sizeof(float)*CpMjJjtGOeWOzwxpAAQP* 
fcLayer->getOutputTensor()->getChannels())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 
fcLayer->getOutputTensor()->getBatchSize(),fcLayer->getOutputTensor()->getChannels(), 
1, 1)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(NMMfJylfQjiIUAKhXCJb, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 
fcLayer->getOutputTensor()->getChannels(), 1, 1)); 
CUDA_CALL(hipMalloc((void**)&MNuwXDSoGEYeABeVTwOh, sizeof(float)*fcLayer->getOutputTensor()->getChannels()));
#ifdef RANDOM
 hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, vjDFlBZzKvbpPseAtMBP, 
fcLayer->getInputTensor()->getChannels()*fcLayer->getInputTensor()->getWidth()*fcLayer->getInputTensor()->getHeight()*fcLayer->getOutputTensor()->getChannels(), 
0, 0.1); hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, MNuwXDSoGEYeABeVTwOh, 
fcLayer->getOutputTensor()->getChannels(), -0.5, 1);
#endif
 int fhikqqlnUKCjleVKDqiG = CpMjJjtGOeWOzwxpAAQP*opTensor->getChannels();  
loadWeights(fhikqqlnUKCjleVKDqiG, wMySyzzledUmSLTWhuYH); 
loadBias(NZjOkZPwLzQsdEVkwMcX); return; } void MWFCLayerImpl::loadWeights(int 
fhikqqlnUKCjleVKDqiG, const char* UKtMXCCqdjeyaVHabkxg) {  MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); FILE* UzaGmBLFEwmwaFXebUma = 
MWCNNLayer::openBinaryFile(UKtMXCCqdjeyaVHabkxg); assert(UzaGmBLFEwmwaFXebUma); float* 
OKaRVOctKLlnIyGmjRNW = MALLOC_CALL(sizeof(float)*fhikqqlnUKCjleVKDqiG); 
fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), fhikqqlnUKCjleVKDqiG, UzaGmBLFEwmwaFXebUma); if( 
ipTensor->getHeight() != 1 && ipTensor->getWidth() != 1 ) { float* 
ONvcEjLBnVNUdjMKOAwF = 
MALLOC_CALL(sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()); for(int 
k=0; k<fhikqqlnUKCjleVKDqiG/ipTensor->getHeight()/ipTensor->getWidth(); k++) { 
for(int i=0; i<ipTensor->getHeight()*ipTensor->getWidth(); i++) 
ONvcEjLBnVNUdjMKOAwF[i]=OKaRVOctKLlnIyGmjRNW[k*ipTensor->getHeight()*ipTensor->getWidth()+i]; 
for(int j=0; j<ipTensor->getHeight(); j++) for(int i=0; i<ipTensor->getWidth(); 
i++) 
OKaRVOctKLlnIyGmjRNW[k*ipTensor->getHeight()*ipTensor->getWidth()+j*ipTensor->getWidth()+i]=ONvcEjLBnVNUdjMKOAwF[j+i*ipTensor->getHeight()]; 
} free(ONvcEjLBnVNUdjMKOAwF); } CUDA_CALL(hipMemcpy(vjDFlBZzKvbpPseAtMBP, 
OKaRVOctKLlnIyGmjRNW, sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", UKtMXCCqdjeyaVHabkxg, fhikqqlnUKCjleVKDqiG, OKaRVOctKLlnIyGmjRNW[0]);
#endif
 free(OKaRVOctKLlnIyGmjRNW); fclose(UzaGmBLFEwmwaFXebUma); return; } void 
MWFCLayerImpl::loadBias(const char* UKtMXCCqdjeyaVHabkxg) { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* opTensor = 
fcLayer->getOutputTensor(0); FILE* UzaGmBLFEwmwaFXebUma = 
MWCNNLayer::openBinaryFile(UKtMXCCqdjeyaVHabkxg); assert(UzaGmBLFEwmwaFXebUma); int 
fhikqqlnUKCjleVKDqiG = opTensor->getChannels();  float* OKaRVOctKLlnIyGmjRNW = 
MALLOC_CALL(sizeof(float)*fhikqqlnUKCjleVKDqiG); fread(OKaRVOctKLlnIyGmjRNW, 
sizeof(float), fhikqqlnUKCjleVKDqiG, UzaGmBLFEwmwaFXebUma); 
CUDA_CALL(hipMemcpy(MNuwXDSoGEYeABeVTwOh, OKaRVOctKLlnIyGmjRNW, 
sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice)); 
free(OKaRVOctKLlnIyGmjRNW); fclose(UzaGmBLFEwmwaFXebUma); return; } void 
MWFCLayerImpl::predict() { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); MWTensor* opTensor = fcLayer->getOutputTensor(0); 
int CpMjJjtGOeWOzwxpAAQP = 
ipTensor->getChannels()*ipTensor->getHeight()*ipTensor->getWidth(); int 
DSsxcjIrUgZCKZovyNQf = opTensor->getChannels(); int YgcpEBUCwCLaPhyntIio=1; 
int ZCArwzdUdwQuFQUWjnUE=1; if( opTensor->getBatchSize()==1 ) { 
CUDA_CALL(hipMemcpy(getData(), MNuwXDSoGEYeABeVTwOh, 
sizeof(float)*DSsxcjIrUgZCKZovyNQf, hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*fYaOQTeunPwVjnhhTECh->getCublasHandle(), HIPBLAS_OP_T, 
CpMjJjtGOeWOzwxpAAQP, DSsxcjIrUgZCKZovyNQf, getOnePtr(), 
vjDFlBZzKvbpPseAtMBP, CpMjJjtGOeWOzwxpAAQP, ipTensor->getData(), 
YgcpEBUCwCLaPhyntIio, getOnePtr(),getData(), ZCArwzdUdwQuFQUWjnUE)); } else { 
CUBLAS_CALL(hipblasSgemm(*fYaOQTeunPwVjnhhTECh->getCublasHandle(), HIPBLAS_OP_T, 
HIPBLAS_OP_N, DSsxcjIrUgZCKZovyNQf, opTensor->getBatchSize(), 
CpMjJjtGOeWOzwxpAAQP, getOnePtr(), vjDFlBZzKvbpPseAtMBP, 
CpMjJjtGOeWOzwxpAAQP, ipTensor->getData(), CpMjJjtGOeWOzwxpAAQP, 
getZeroPtr(),getData(), DSsxcjIrUgZCKZovyNQf)); 
CUDNN_CALL(hipdnnAddTensor(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), getOnePtr(), 
NMMfJylfQjiIUAKhXCJb, MNuwXDSoGEYeABeVTwOh, getOnePtr(), 
*getOutputDescriptor(),getData())); } return; } void MWFCLayerImpl::cleanup() { 
if (vjDFlBZzKvbpPseAtMBP) { call_cuda_free(vjDFlBZzKvbpPseAtMBP); }  
CUDNN_CALL(hipdnnDestroyTensorDescriptor(NMMfJylfQjiIUAKhXCJb)); if 
(MNuwXDSoGEYeABeVTwOh) { call_cuda_free(MNuwXDSoGEYeABeVTwOh); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) call_cuda_free(data); 
} } } MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int outbufIdx) : MWCNNLayerImpl(layer, 
ntwk_impl)  {  CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createSoftmaxLayer(outbufIdx); } MWSoftmaxLayerImpl::~MWSoftmaxLayerImpl() { } 
void MWSoftmaxLayerImpl::createSoftmaxLayer(int outbufIdx) { MWSoftmaxLayer* 
sfmxLayer = static_cast<MWSoftmaxLayer*>(getLayer()); MWTensor* ipTensor = 
sfmxLayer->getInputTensor(0); MWTensor* opTensor = 
sfmxLayer->getOutputTensor(0); int numOutputFeatures = ipTensor->getChannels(); 
if (outbufIdx < 0) { CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()*numOutputFeatures*ipTensor->getBatchSize())); 
} else { setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth()));  
return; } void MWSoftmaxLayerImpl::predict() { MWSoftmaxLayer* sfmxLayer = 
static_cast<MWSoftmaxLayer*>(getLayer()); MWTensor* ipTensor = 
sfmxLayer->getInputTensor(0); MWTensor* opTensor = 
sfmxLayer->getOutputTensor(0); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(ipTensor);  
CUDNN_CALL(hipdnnSoftmaxForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), ipDesc, 
ipTensor->getData(), getZeroPtr(), *getOutputDescriptor(), getData())); } void 
MWSoftmaxLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) call_cuda_free(data); 
} } } MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int GFienSVKLlDQuZeqAdLC,  int GeeOVBfQrpMacIFBLKOo,  
int GsZlHFuhbvjLtRMDjXnW,  int HJHXkKmgFxxIOsIvRRnF,  int DqxLTLaJwwgQqmrtCDuu,  int 
FeVcBgtQmTLtmnNcJGMY, int outbufIdx) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&npGnQZLrEfVTQnEbwqij)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createAvgPoolingLayer(GFienSVKLlDQuZeqAdLC, GeeOVBfQrpMacIFBLKOo, GsZlHFuhbvjLtRMDjXnW, 
HJHXkKmgFxxIOsIvRRnF, DqxLTLaJwwgQqmrtCDuu, FeVcBgtQmTLtmnNcJGMY, outbufIdx); } 
MWAvgPoolingLayerImpl::~MWAvgPoolingLayerImpl() { } void 
MWAvgPoolingLayerImpl::createAvgPoolingLayer(int GFienSVKLlDQuZeqAdLC, int 
GeeOVBfQrpMacIFBLKOo, int GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, int 
DqxLTLaJwwgQqmrtCDuu, int FeVcBgtQmTLtmnNcJGMY, int outbufIdx) { 
MWAvgPoolingLayer* avgpoolLayer = static_cast<MWAvgPoolingLayer*>(getLayer()); 
MWTensor* ipTensor = avgpoolLayer->getInputTensor(0); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(npGnQZLrEfVTQnEbwqij, 
HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, HIPDNN_NOT_PROPAGATE_NAN, 
GFienSVKLlDQuZeqAdLC, GeeOVBfQrpMacIFBLKOo, DqxLTLaJwwgQqmrtCDuu, FeVcBgtQmTLtmnNcJGMY, 
GsZlHFuhbvjLtRMDjXnW, HJHXkKmgFxxIOsIvRRnF)); int fSKMHAqIghbYYgyIpNDw, OumvfgWXDdmsQaciHMHx, 
WprSrhAStKGxyXeoxETy, vjDFlBZzKvbpPseAtMBP;  hipdnnTensorDescriptor_t bUVPfnrJhLfHzOLUUrKk = 
*getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(npGnQZLrEfVTQnEbwqij, 
bUVPfnrJhLfHzOLUUrKk, &fSKMHAqIghbYYgyIpNDw ,&OumvfgWXDdmsQaciHMHx, &WprSrhAStKGxyXeoxETy, 
&vjDFlBZzKvbpPseAtMBP)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fSKMHAqIghbYYgyIpNDw, OumvfgWXDdmsQaciHMHx, WprSrhAStKGxyXeoxETy, 
vjDFlBZzKvbpPseAtMBP)); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*fSKMHAqIghbYYgyIpNDw*OumvfgWXDdmsQaciHMHx*WprSrhAStKGxyXeoxETy*vjDFlBZzKvbpPseAtMBP)); } else { 
setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
avgpoolLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } } void 
MWAvgPoolingLayerImpl::predict() { MWAvgPoolingLayer* avgpoolLayer = 
static_cast<MWAvgPoolingLayer*>(getLayer()); MWTensor* ipTensor = 
avgpoolLayer->getInputTensor(0); MWTensor* opTensor = 
avgpoolLayer->getOutputTensor(0); hipdnnTensorDescriptor_t bUVPfnrJhLfHzOLUUrKk = 
*getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnPoolingForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
npGnQZLrEfVTQnEbwqij, getOnePtr(), bUVPfnrJhLfHzOLUUrKk, ipTensor->getData(), 
getZeroPtr(), *getOutputDescriptor(),opTensor->getData())); } void 
MWAvgPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(npGnQZLrEfVTQnEbwqij)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) call_cuda_free(data); 
} } } MWOutputLayerImpl::MWOutputLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int ) : MWCNNLayerImpl(layer, ntwk_impl) { 
createOutputLayer(); } MWOutputLayerImpl::~MWOutputLayerImpl() { } void 
MWOutputLayerImpl::createOutputLayer() { MWOutputLayer* opLayer = 
static_cast<MWOutputLayer*>(getLayer()); MWTensor* ipTensor = 
opLayer->getInputTensor(0); setData(ipTensor->getData()); return; } void 
MWOutputLayerImpl::predict() { } void MWOutputLayerImpl::cleanup() { }