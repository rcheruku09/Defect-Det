//
// File: segnet_predict_initialize.cu
//
// GPU Coder version                    : 1.2
// CUDA/C/C++ source code generated on  : 02-Nov-2018 17:52:01
//

// Include Files
#include "segnet_predict.h"
#include "segnet_predict_initialize.h"

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
void segnet_predict_initialize()
{
  hipSetDevice(0);
  segnet_predict_init();
}

//
// File trailer for segnet_predict_initialize.cu
//
// [EOF]
//
