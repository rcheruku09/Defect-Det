#include "hip/hip_runtime.h"
//
// File: predict.cu
//
// GPU Coder version                    : 1.2
// CUDA/C/C++ source code generated on  : 02-Nov-2018 17:52:01
//

// Include Files
#include "MWCudaDimUtility.h"
#include "segnet_predict.h"
#include "predict.h"
#include "DeepLearningNetwork.h"

// Function Declarations
static __global__ void c_DeepLearningNetwork_predict_k(const uint8_T inputdata
  [518400], uint8_T b_inputdata[518400]);
static __global__ void d_DeepLearningNetwork_predict_k(uint8_T inputdata[518400],
  real32_T inputT[518400]);
static __global__ void e_DeepLearningNetwork_predict_k(real32_T out[1900800],
  real32_T outT[1900800]);

// Function Definitions

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const uint8_T inputdata[518400]
//                uint8_T b_inputdata[518400]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void c_DeepLearningNetwork_predict_k
  (const uint8_T inputdata[518400], uint8_T b_inputdata[518400])
{
  uint32_T threadId;
  int32_T i0;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  i0 = (int32_T)threadId;
  if (i0 < 518400) {
    b_inputdata[i0] = inputdata[i0];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                uint8_T inputdata[518400]
//                real32_T inputT[518400]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void d_DeepLearningNetwork_predict_k
  (uint8_T inputdata[518400], real32_T inputT[518400])
{
  uint32_T threadId;
  int32_T i0;
  int32_T i1;
  int32_T p;
  uint32_T tmpIndex;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  i0 = (int32_T)(threadId % 480U);
  tmpIndex = (threadId - (uint32_T)i0) / 480U;
  i1 = (int32_T)(tmpIndex % 360U);
  tmpIndex = (tmpIndex - (uint32_T)i1) / 360U;
  p = (int32_T)tmpIndex;
  if (p < 3) {
    inputT[(i0 + 480 * i1) + 172800 * p] = (real32_T)inputdata[(i1 + 360 * i0) +
      172800 * p];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                real32_T out[1900800]
//                real32_T outT[1900800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void e_DeepLearningNetwork_predict_k
  (real32_T out[1900800], real32_T outT[1900800])
{
  uint32_T threadId;
  int32_T i0;
  int32_T i1;
  int32_T p;
  uint32_T tmpIndex;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  i0 = (int32_T)(threadId % 360U);
  tmpIndex = (threadId - (uint32_T)i0) / 360U;
  i1 = (int32_T)(tmpIndex % 480U);
  tmpIndex = (tmpIndex - (uint32_T)i1) / 480U;
  p = (int32_T)tmpIndex;
  if (p < 11) {
    outT[(i0 + 360 * i1) + 172800 * p] = out[(i1 + 480 * i0) + 172800 * p];
  }
}

//
// Arguments    : b_SegNet_0 *obj
//                const uint8_T inputdata[518400]
//                real32_T outT[1900800]
// Return Type  : void
//
void DeepLearningNetwork_predict(b_SegNet_0 *obj, const uint8_T inputdata[518400],
  real32_T outT[1900800])
{
  real32_T (*gpu_inputT)[518400];
  real32_T (*gpu_out)[1900800];
  uint8_T (*gpu_inputdata)[518400];
  uint8_T (*b_gpu_inputdata)[518400];
  real32_T (*gpu_outT)[1900800];
  hipMalloc(&gpu_outT, 7603200UL);
  hipMalloc(&gpu_out, 7603200UL);
  hipMalloc(&gpu_inputT, 2073600UL);
  hipMalloc(&b_gpu_inputdata, 518400UL);
  hipMalloc(&gpu_inputdata, 518400UL);
  hipMemcpy(gpu_inputdata, (void *)&inputdata[0], 518400UL,
             hipMemcpyHostToDevice);
  c_DeepLearningNetwork_predict_k<<<dim3(1013U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_inputdata, *b_gpu_inputdata);
  d_DeepLearningNetwork_predict_k<<<dim3(1013U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*b_gpu_inputdata, *gpu_inputT);
  hipMemcpy(obj->inputData, *gpu_inputT, 518400UL * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy(*gpu_out, obj->outputData, 1900800UL * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  e_DeepLearningNetwork_predict_k<<<dim3(3713U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_out, *gpu_outT);
  hipMemcpy(&outT[0], gpu_outT, 7603200UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_inputdata);
  hipFree(*b_gpu_inputdata);
  hipFree(*gpu_inputT);
  hipFree(*gpu_out);
  hipFree(*gpu_outT);
}

//
// File trailer for predict.cu
//
// [EOF]
//
