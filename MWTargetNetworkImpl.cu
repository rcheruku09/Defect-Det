#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWCNNLayerImpl.hpp"
 void MWTargetNetworkImpl::preSetup(int BufSize,int numBufsToAlloc) { numBufs = 
numBufsToAlloc; for(int i = 0; i < numBufs; i++) { float *memPtr ; 
hipMalloc((void**)&memPtr, sizeof(float)*BufSize); 
memBuffer.push_back(memPtr); } QjgQHaUACFNSteMrRtRj = new hipblasHandle_t; 
hipblasCreate(QjgQHaUACFNSteMrRtRj); QwUuNuQNtlPXrIwRNiSZ = new 
hipdnnHandle_t; hipdnnCreate(QwUuNuQNtlPXrIwRNiSZ); } void 
MWTargetNetworkImpl::postSetup(MWCNNLayer* layers[],int numLayers) { 
createWorkSpace(&xkUNToJIgvoLoUQuzKRF); for (int i = 0; i < numLayers; i++) { 
layers[i]->postSetup();  } if ((GnxRkpzrPZimKtYYHSuG != 
*getWorkSpaceSize() && GnxRkpzrPZimKtYYHSuG > 0)) { 
setWorkSpaceSize(GnxRkpzrPZimKtYYHSuG); if (xkUNToJIgvoLoUQuzKRF) 
{ hipFree(xkUNToJIgvoLoUQuzKRF); xkUNToJIgvoLoUQuzKRF = 0; } 
CUDA_CALL(hipMalloc((void**)&xkUNToJIgvoLoUQuzKRF, *getWorkSpaceSize())); }  
if (!xkUNToJIgvoLoUQuzKRF && (*getWorkSpaceSize() > 0)) { throw 
std::runtime_error("Out of memory. Unable to allocate workspace."); } } void 
MWTargetNetworkImpl::createWorkSpace(float** yCdIUfwoZFngCRRRkCTg) { 
hipError_t qWwjVYwfnvEnFKlgpqwA = hipMalloc((void**)yCdIUfwoZFngCRRRkCTg, 
omxlPZbBePZdWaJOBUUG); if (qWwjVYwfnvEnFKlgpqwA != hipSuccess) { 
*yCdIUfwoZFngCRRRkCTg = 0;  } } void 
MWTargetNetworkImpl::setWorkSpaceSize(size_t wss) { omxlPZbBePZdWaJOBUUG 
= wss;  } size_t* MWTargetNetworkImpl::getWorkSpaceSize() { return 
&omxlPZbBePZdWaJOBUUG; } float* MWTargetNetworkImpl::getWorkSpace() { 
return xkUNToJIgvoLoUQuzKRF; } size_t* 
MWTargetNetworkImpl::getPostSetupWorkSpaceSize() { return 
&GnxRkpzrPZimKtYYHSuG; } void 
MWTargetNetworkImpl::setPostSetupWorkSpaceSize(size_t psWSize) { 
GnxRkpzrPZimKtYYHSuG = psWSize; } hipblasHandle_t* 
MWTargetNetworkImpl::getCublasHandle() { return QjgQHaUACFNSteMrRtRj; } 
hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
QwUuNuQNtlPXrIwRNiSZ; } void MWTargetNetworkImpl::setAutoTune(bool 
autotune) { MW_autoTune = autotune; } bool MWTargetNetworkImpl::getAutoTune() 
const { return MW_autoTune; } void MWTargetNetworkImpl::cleanup() { if 
(xkUNToJIgvoLoUQuzKRF) { hipFree(xkUNToJIgvoLoUQuzKRF); } if 
(QjgQHaUACFNSteMrRtRj) { hipblasDestroy(*QjgQHaUACFNSteMrRtRj); } if 
(QwUuNuQNtlPXrIwRNiSZ) { hipdnnDestroy(*QwUuNuQNtlPXrIwRNiSZ); } for(int 
i = 0; i < numBufs; i++) { float *memPtr = memBuffer[i]; hipError_t 
qWwjVYwfnvEnFKlgpqwA = hipFree(memPtr); if (qWwjVYwfnvEnFKlgpqwA != 
hipErrorDeinitialized) { CUDA_CALL(qWwjVYwfnvEnFKlgpqwA); } } }