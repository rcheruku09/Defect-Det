#include "hip/hip_runtime.h"
#include "MWMaxUnpoolingLayerImpl.hpp"
#include "MWMaxUnpoolingLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include <cassert>
 MWMaxUnpoolingLayerImpl::MWMaxUnpoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int outbufIdx) : MWCNNLayerImpl(layer, 
ntwk_impl) { createUnpoolingLayer(outbufIdx); } 
MWMaxUnpoolingLayerImpl::~MWMaxUnpoolingLayerImpl() {  } void 
MWMaxUnpoolingLayerImpl::createUnpoolingLayer(int outbufIdx) { MWTensor* 
opTensor = getLayer()->getOutputTensor(0); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, 
sizeof(float)*opTensor->getBatchSize()* opTensor->getChannels()* 
opTensor->getHeight()* opTensor->getWidth())); } else { 
setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } CUDA_CALL(hipMemset(getData(),0.0f, 
sizeof(float)*opTensor->getBatchSize()* opTensor->getChannels()* 
opTensor->getHeight()* opTensor->getWidth() ));  
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
MWMaxUnpoolingLayerImpl::predict() { assert(this->getData() != 
getLayer()->getInputTensor(0)->getData()); 
doMaxUnpoolingForwardImpl(getLayer()->getInputTensor(0)->getData(), 
getLayer()->getInputTensor(1)->getData(), 
getLayer()->getOutputTensor(0)->getData(), 
getLayer()->getInputTensor(0)->getHeight(), 
getLayer()->getInputTensor(0)->getWidth(), 
getLayer()->getInputTensor(0)->getChannels(), 
getLayer()->getInputTensor(0)->getBatchSize()); return; } void __global__ 
__launch_bounds__(1024) MaxUnpoolingImpl(float * inputBuffer, float * 
indexBuffer, float * outputBuffer, const int CDJtexcMbXMWAmnNZsNf) { for(int i = 
blockDim.x * blockIdx.x + threadIdx.x; i < CDJtexcMbXMWAmnNZsNf; i+= 
blockDim.x*gridDim.x) { outputBuffer[static_cast<int>(indexBuffer[i])] = 
inputBuffer[i]; } } void 
MWMaxUnpoolingLayerImpl::doMaxUnpoolingForwardImpl(float* inputBuffer, float* 
indexBuffer, float* outputBuffer, int XCLDbxHBtWRStETWIkId, int wtNPjzxHKNoJIigzXrEl, 
int hDaNSVZAofAENeIAiWEw, int MdSWZSOAjugbWppryHbR ) {  
CUDA_CALL(hipMemset(outputBuffer,0, 
sizeof(float)*getLayer()->getOutputTensor(0)->getBatchSize()* 
getLayer()->getOutputTensor(0)->getChannels()* 
getLayer()->getOutputTensor(0)->getHeight()* 
getLayer()->getOutputTensor(0)->getWidth() )); int fjfzkUfcCOqjrkAVGfuc = 
XCLDbxHBtWRStETWIkId*wtNPjzxHKNoJIigzXrEl* 
hDaNSVZAofAENeIAiWEw*MdSWZSOAjugbWppryHbR; int 
tqZLvfMHdgZzbchUyDzd = (fjfzkUfcCOqjrkAVGfuc < 1024) ? fjfzkUfcCOqjrkAVGfuc : 
1024; int NldNILHvuQqQPSAHXxdT = (fjfzkUfcCOqjrkAVGfuc + 
tqZLvfMHdgZzbchUyDzd - 1)/tqZLvfMHdgZzbchUyDzd; 
MaxUnpoolingImpl<<<NldNILHvuQqQPSAHXxdT, tqZLvfMHdgZzbchUyDzd>>>( 
inputBuffer, indexBuffer, outputBuffer, fjfzkUfcCOqjrkAVGfuc); } void 
MWMaxUnpoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) { float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) call_cuda_free(data); 
} } }