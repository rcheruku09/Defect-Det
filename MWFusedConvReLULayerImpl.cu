#include "MWFusedConvReLULayer.hpp"
#include "MWFusedConvReLULayerImpl.hpp"
#include <cassert>
#include <stdio.h>
 MWFusedConvReLULayerImpl::MWFusedConvReLULayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, int 
EvebzoroiuKkIxwjkGnD, int ECTnqgWHyHCHCLBZlffd, int FrpxvsDMwwgbpqHXWxmN, 
int FwLnexHgxHRquTKmNpoa, int ATEikvMQPqBefhJzjzhc, int 
AwZQzUhuWVLGrWgLHRuM, const char* wMySyzzledUmSLTWhuYH, const char* 
NZjOkZPwLzQsdEVkwMcX, int outbufIdx) : MWCNNLayerImpl(layer, ntwk_impl) , 
zzWugmJRYlNEuAzHMpeQ(NULL) , vjDFlBZzKvbpPseAtMBP(NULL) , MNuwXDSoGEYeABeVTwOh(NULL) , 
vxtNGOWYjhKeBBSzuIMB(NULL) , MUmglsoWcEiRiAZsclur(NULL) , XLJXOFXdnZOyJvtltbyr(NULL) , 
aLsOwwcceEmRSYzllBNs(NULL) , cQBKlCKXxecGPJrXBXdk(0) , 
AzTsxYcYjIEJsGQbeYHm(filt_H) , BLjrjqvCcCommiXWQLjs (filt_W) , 
ClEhcJFlvGCgiavziIag (numGrps) , CGbFsczkgkhjcHoCKzBx (numChnls) , 
CZNYmBcNFSZWvaCklqeM (numFilts) { fYaOQTeunPwVjnhhTECh = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&QMgBqCuvjnbWHWiVPEwn)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&UpnEytIWGokwbTFkBcSx)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&NMMfJylfQjiIUAKhXCJb)); 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&olKGEIcsxmLSoMhRhEtP)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createFusedConvReLULayer(GsZlHFuhbvjLtRMDjXnW, HJHXkKmgFxxIOsIvRRnF, 
EvebzoroiuKkIxwjkGnD, ECTnqgWHyHCHCLBZlffd, FrpxvsDMwwgbpqHXWxmN, 
FwLnexHgxHRquTKmNpoa, ATEikvMQPqBefhJzjzhc, AwZQzUhuWVLGrWgLHRuM, 
wMySyzzledUmSLTWhuYH, NZjOkZPwLzQsdEVkwMcX, outbufIdx); } 
MWFusedConvReLULayerImpl::~MWFusedConvReLULayerImpl() { } float 
MWFusedConvReLULayerImpl::getIsGrouped() { return cQBKlCKXxecGPJrXBXdk; } void 
MWFusedConvReLULayerImpl::setIsGrouped(float ig) { cQBKlCKXxecGPJrXBXdk = ig; 
return; } void MWFusedConvReLULayerImpl::setOutput2(float* out2) { 
zzWugmJRYlNEuAzHMpeQ = out2; return; } float* MWFusedConvReLULayerImpl::getOutput2() { 
return zzWugmJRYlNEuAzHMpeQ; } hipdnnTensorDescriptor_t* 
MWFusedConvReLULayerImpl::getGroupDescriptor() { return &WOJynDmqVUPWjAGVIuMQ; 
} void MWFusedConvReLULayerImpl::createFusedConvReLULayer(int 
GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, int EvebzoroiuKkIxwjkGnD, int 
ECTnqgWHyHCHCLBZlffd , int FrpxvsDMwwgbpqHXWxmN, int FwLnexHgxHRquTKmNpoa, 
int ATEikvMQPqBefhJzjzhc, int AwZQzUhuWVLGrWgLHRuM, const char* 
wMySyzzledUmSLTWhuYH, const char* NZjOkZPwLzQsdEVkwMcX, int outbufIdx) { 
MWTensor* ipTensor = getLayer()->getInputTensor(0); int 
QVgVGfoCXYiYXzPhvVPX = EvebzoroiuKkIxwjkGnD; int 
QhTesEEIHwhNmHSeYbRR = FrpxvsDMwwgbpqHXWxmN; if 
((EvebzoroiuKkIxwjkGnD != ECTnqgWHyHCHCLBZlffd) || (FrpxvsDMwwgbpqHXWxmN != 
FwLnexHgxHRquTKmNpoa)) { float* newInput; int inputH = ipTensor->getHeight() + 
EvebzoroiuKkIxwjkGnD + ECTnqgWHyHCHCLBZlffd; int inputW = 
ipTensor->getWidth() + FrpxvsDMwwgbpqHXWxmN + FwLnexHgxHRquTKmNpoa; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*ipTensor->getBatchSize() 
* ipTensor->getChannels() * inputH * inputW)); CUDA_CALL(hipMemset(newInput, 
0, 
sizeof(float)*ipTensor->getBatchSize()*ipTensor->getChannels()*inputH*inputW)); 
XLJXOFXdnZOyJvtltbyr = new MWTensor(inputH, inputW, ipTensor->getChannels(), 
ipTensor->getBatchSize(), newInput,getLayer(), 0); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bUVPfnrJhLfHzOLUUrKk)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(bUVPfnrJhLfHzOLUUrKk, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, XLJXOFXdnZOyJvtltbyr->getBatchSize(), XLJXOFXdnZOyJvtltbyr->getChannels(), 
XLJXOFXdnZOyJvtltbyr->getHeight(), XLJXOFXdnZOyJvtltbyr->getWidth())); 
QVgVGfoCXYiYXzPhvVPX = 0; QhTesEEIHwhNmHSeYbRR = 0; } else { 
XLJXOFXdnZOyJvtltbyr = ipTensor; bUVPfnrJhLfHzOLUUrKk = 
*getCuDNNDescriptor(XLJXOFXdnZOyJvtltbyr); } eVAFqeShtGZAZluKdMvQ = 
EvebzoroiuKkIxwjkGnD; eqOmMKQRpqBqRQCnJmxt = FrpxvsDMwwgbpqHXWxmN; 
assert(XLJXOFXdnZOyJvtltbyr != NULL); MWFusedConvReLULayer* fusedConvReluLayer = static_cast<MWFusedConvReLULayer*>(getLayer());
#if (CUDNN_MAJOR <= 5)
 { if ((ATEikvMQPqBefhJzjzhc != 1) && (AwZQzUhuWVLGrWgLHRuM != 1)){ 
printf("Dilated Convolution only supported for cuDNN 6 or greater "); throw 
std::runtime_error("Unsupported Dilation Factor"); } 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(QMgBqCuvjnbWHWiVPEwn, 
QVgVGfoCXYiYXzPhvVPX, QhTesEEIHwhNmHSeYbRR, GsZlHFuhbvjLtRMDjXnW, 
HJHXkKmgFxxIOsIvRRnF, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(QMgBqCuvjnbWHWiVPEwn, 
QVgVGfoCXYiYXzPhvVPX, QhTesEEIHwhNmHSeYbRR, GsZlHFuhbvjLtRMDjXnW, 
HJHXkKmgFxxIOsIvRRnF, ATEikvMQPqBefhJzjzhc, AwZQzUhuWVLGrWgLHRuM, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT)); }
#endif
 CUDNN_CALL(hipdnnSetActivationDescriptor(olKGEIcsxmLSoMhRhEtP, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); int sxuOMwKXOKfuExclRaSe, 
sRECVoNNtDdcBOWgDyar; int numInputFeatures = 
CGbFsczkgkhjcHoCKzBx*ClEhcJFlvGCgiavziIag; int 
hnewnpwgzKmOdualajhn,shEncNmxJsMuJKwbrwok,tnTPxeDjBsqLAPkJcPJX; MWTensor* 
opTensor = fusedConvReluLayer->getOutputTensor(0); hnewnpwgzKmOdualajhn 
= opTensor->getChannels(); shEncNmxJsMuJKwbrwok = opTensor->getHeight(); 
tnTPxeDjBsqLAPkJcPJX = opTensor->getWidth(); size_t ugnnrhsgTeWucrMPCJUc = 0; if( 
ClEhcJFlvGCgiavziIag == 1 ) { 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(UpnEytIWGokwbTFkBcSx, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, hnewnpwgzKmOdualajhn, numInputFeatures, 
AzTsxYcYjIEJsGQbeYHm, BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(NMMfJylfQjiIUAKhXCJb, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, hnewnpwgzKmOdualajhn, 1, 1)); 
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(QMgBqCuvjnbWHWiVPEwn, 
bUVPfnrJhLfHzOLUUrKk, UpnEytIWGokwbTFkBcSx, &sxuOMwKXOKfuExclRaSe, 
&sRECVoNNtDdcBOWgDyar, &shEncNmxJsMuJKwbrwok, &tnTPxeDjBsqLAPkJcPJX)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, sxuOMwKXOKfuExclRaSe, sRECVoNNtDdcBOWgDyar, 
opTensor->getHeight(), opTensor->getWidth())); assert(opTensor->getHeight() == 
shEncNmxJsMuJKwbrwok); assert(opTensor->getWidth() == tnTPxeDjBsqLAPkJcPJX);
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
bUVPfnrJhLfHzOLUUrKk, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, 
*getOutputDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, 
&PmFfARVzoHVAYkfpuvqK)); }
#else
 { hipdnnConvolutionFwdAlgoPerf_t perf_results[3]; int returnedAlgoCount; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
bUVPfnrJhLfHzOLUUrKk, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, 
*getOutputDescriptor(), 3, &returnedAlgoCount, perf_results)); 
PmFfARVzoHVAYkfpuvqK = perf_results[0].algo; }
#endif
 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
bUVPfnrJhLfHzOLUUrKk, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, 
*getOutputDescriptor(), PmFfARVzoHVAYkfpuvqK, &ugnnrhsgTeWucrMPCJUc)); } else { 
setIsGrouped(1); MWTensor* ipTensor = XLJXOFXdnZOyJvtltbyr; aLsOwwcceEmRSYzllBNs = 
ipTensor->getData() + ipTensor->getChannels()/ClEhcJFlvGCgiavziIag * 
ipTensor->getHeight() * ipTensor->getWidth(); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&cCXqPFPPcoHzYMDpnUxQ)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(cCXqPFPPcoHzYMDpnUxQ, 
HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(), 
ipTensor->getChannels()/ClEhcJFlvGCgiavziIag, ipTensor->getHeight(), 
ipTensor->getWidth(), 
ipTensor->getChannels()*ipTensor->getHeight()*ipTensor->getWidth(), 
ipTensor->getHeight()*ipTensor->getWidth(), ipTensor->getWidth(), 1)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getGroupDescriptor())); 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(UpnEytIWGokwbTFkBcSx, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, CZNYmBcNFSZWvaCklqeM, CGbFsczkgkhjcHoCKzBx, 
AzTsxYcYjIEJsGQbeYHm, BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(QMgBqCuvjnbWHWiVPEwn, 
cCXqPFPPcoHzYMDpnUxQ, UpnEytIWGokwbTFkBcSx, &sxuOMwKXOKfuExclRaSe, 
&sRECVoNNtDdcBOWgDyar, &shEncNmxJsMuJKwbrwok, &tnTPxeDjBsqLAPkJcPJX)); 
assert(opTensor->getHeight() == shEncNmxJsMuJKwbrwok); assert(opTensor->getWidth() 
== tnTPxeDjBsqLAPkJcPJX); 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(*getGroupDescriptor(), 
HIPDNN_DATA_FLOAT, sxuOMwKXOKfuExclRaSe, sRECVoNNtDdcBOWgDyar, shEncNmxJsMuJKwbrwok, 
tnTPxeDjBsqLAPkJcPJX, 
sRECVoNNtDdcBOWgDyar*ClEhcJFlvGCgiavziIag*shEncNmxJsMuJKwbrwok*tnTPxeDjBsqLAPkJcPJX, 
shEncNmxJsMuJKwbrwok*tnTPxeDjBsqLAPkJcPJX, tnTPxeDjBsqLAPkJcPJX, 1)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, sxuOMwKXOKfuExclRaSe, 
sRECVoNNtDdcBOWgDyar*ClEhcJFlvGCgiavziIag, shEncNmxJsMuJKwbrwok, tnTPxeDjBsqLAPkJcPJX)); 
assert(CZNYmBcNFSZWvaCklqeM == sRECVoNNtDdcBOWgDyar); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(NMMfJylfQjiIUAKhXCJb, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, sRECVoNNtDdcBOWgDyar, 1, 1));
#if (CUDNN_MAJOR < 7)
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
cCXqPFPPcoHzYMDpnUxQ, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, 
*getGroupDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &PmFfARVzoHVAYkfpuvqK));
#else
 hipdnnConvolutionFwdAlgoPerf_t perf_results[3]; int returnedAlgoCount; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
cCXqPFPPcoHzYMDpnUxQ, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, 
*getGroupDescriptor(), 3, &returnedAlgoCount,perf_results)); 
PmFfARVzoHVAYkfpuvqK = perf_results[0].algo;
#endif
 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
cCXqPFPPcoHzYMDpnUxQ, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, 
*getGroupDescriptor(), PmFfARVzoHVAYkfpuvqK, &ugnnrhsgTeWucrMPCJUc)); } if( 
ugnnrhsgTeWucrMPCJUc > *fYaOQTeunPwVjnhhTECh->getWorkSpaceSize() ) { 
fYaOQTeunPwVjnhhTECh->setWorkSpaceSize(ugnnrhsgTeWucrMPCJUc); } 
assert(sxuOMwKXOKfuExclRaSe == ipTensor->getBatchSize()); 
assert(hnewnpwgzKmOdualajhn == sRECVoNNtDdcBOWgDyar * 
ClEhcJFlvGCgiavziIag); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&RAtlBpdedvgxUsgDTsch, sizeof(float) * 
opTensor->getBatchSize() * opTensor->getChannels() * opTensor->getHeight() * 
opTensor->getWidth())); } else { 
setData(fYaOQTeunPwVjnhhTECh->memBuffer[outbufIdx]); 
getLayer()->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDA_CALL(hipMalloc((void**)&vjDFlBZzKvbpPseAtMBP, 
sizeof(float)*CGbFsczkgkhjcHoCKzBx*hnewnpwgzKmOdualajhn*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs)); 
CUDA_CALL(hipMalloc((void**)&MNuwXDSoGEYeABeVTwOh, sizeof(float)*hnewnpwgzKmOdualajhn));
#ifdef RANDOM
 hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, vjDFlBZzKvbpPseAtMBP, 
CGbFsczkgkhjcHoCKzBx*hnewnpwgzKmOdualajhn*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs, 
0, 0.1); hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, MNuwXDSoGEYeABeVTwOh, 
hnewnpwgzKmOdualajhn, -0.5, 1);
#endif
 if( ClEhcJFlvGCgiavziIag == 2 ) { vxtNGOWYjhKeBBSzuIMB = vjDFlBZzKvbpPseAtMBP + 
CZNYmBcNFSZWvaCklqeM * CGbFsczkgkhjcHoCKzBx * AzTsxYcYjIEJsGQbeYHm * 
BLjrjqvCcCommiXWQLjs; MUmglsoWcEiRiAZsclur = MNuwXDSoGEYeABeVTwOh + CZNYmBcNFSZWvaCklqeM; 
setOutput2(getData() + hnewnpwgzKmOdualajhn/ 2 * shEncNmxJsMuJKwbrwok * 
tnTPxeDjBsqLAPkJcPJX); setIsGrouped(1); } loadWeights(wMySyzzledUmSLTWhuYH); 
loadBias(NZjOkZPwLzQsdEVkwMcX); return; } void 
MWFusedConvReLULayerImpl::predict() { MWFusedConvReLULayer* fusedConvReluLayer 
= static_cast<MWFusedConvReLULayer*>(getLayer()); if (XLJXOFXdnZOyJvtltbyr != 
fusedConvReluLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(XLJXOFXdnZOyJvtltbyr->getData(), 0, 
sizeof(float)*XLJXOFXdnZOyJvtltbyr->getBatchSize()*XLJXOFXdnZOyJvtltbyr->getChannels()*XLJXOFXdnZOyJvtltbyr->getHeight()*XLJXOFXdnZOyJvtltbyr->getWidth())); 
int gzSTokDHvkXefhiGDcWL = 
fusedConvReluLayer->getInputTensor()->getHeight()*fusedConvReluLayer->getInputTensor()->getWidth()*fusedConvReluLayer->getInputTensor()->getBatchSize()*fusedConvReluLayer->getInputTensor()->getChannels(); 
MWCNNLayerImpl::padInput(fusedConvReluLayer->getInputTensor()->getData(), 
fusedConvReluLayer->getInputTensor()->getHeight(), 
fusedConvReluLayer->getInputTensor()->getWidth(), 
fusedConvReluLayer->getInputTensor()->getChannels(), 
XLJXOFXdnZOyJvtltbyr->getHeight(), XLJXOFXdnZOyJvtltbyr->getWidth(), eVAFqeShtGZAZluKdMvQ, 
eqOmMKQRpqBqRQCnJmxt, XLJXOFXdnZOyJvtltbyr->getData(), gzSTokDHvkXefhiGDcWL); } 
if(ClEhcJFlvGCgiavziIag == 1 ) { assert(getData() != XLJXOFXdnZOyJvtltbyr->getData()); 
CUDNN_CALL(cudnnConvolutionBiasActivationForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
getOnePtr(), bUVPfnrJhLfHzOLUUrKk, XLJXOFXdnZOyJvtltbyr->getData(), 
UpnEytIWGokwbTFkBcSx, vjDFlBZzKvbpPseAtMBP, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK, 
fYaOQTeunPwVjnhhTECh->getWorkSpace(), *fYaOQTeunPwVjnhhTECh->getWorkSpaceSize(), 
getZeroPtr(), *getOutputDescriptor(), XLJXOFXdnZOyJvtltbyr->getData(), 
NMMfJylfQjiIUAKhXCJb, MNuwXDSoGEYeABeVTwOh, olKGEIcsxmLSoMhRhEtP, *getOutputDescriptor(), 
getData())); } else { assert(getData() != XLJXOFXdnZOyJvtltbyr->getData()); 
CUDNN_CALL(cudnnConvolutionBiasActivationForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
getOnePtr(), cCXqPFPPcoHzYMDpnUxQ, XLJXOFXdnZOyJvtltbyr->getData(), 
UpnEytIWGokwbTFkBcSx, vjDFlBZzKvbpPseAtMBP, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK, 
fYaOQTeunPwVjnhhTECh->getWorkSpace(), *fYaOQTeunPwVjnhhTECh->getWorkSpaceSize(), 
getZeroPtr(), *getGroupDescriptor(), XLJXOFXdnZOyJvtltbyr->getData(), 
NMMfJylfQjiIUAKhXCJb, MNuwXDSoGEYeABeVTwOh, olKGEIcsxmLSoMhRhEtP, *getGroupDescriptor(), 
getData())); 
CUDNN_CALL(cudnnConvolutionBiasActivationForward(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
getOnePtr(), cCXqPFPPcoHzYMDpnUxQ, aLsOwwcceEmRSYzllBNs, UpnEytIWGokwbTFkBcSx, 
vxtNGOWYjhKeBBSzuIMB, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK, 
fYaOQTeunPwVjnhhTECh->getWorkSpace(), *fYaOQTeunPwVjnhhTECh->getWorkSpaceSize(), 
getZeroPtr(), *getGroupDescriptor(), aLsOwwcceEmRSYzllBNs, NMMfJylfQjiIUAKhXCJb, 
MUmglsoWcEiRiAZsclur, olKGEIcsxmLSoMhRhEtP, *getGroupDescriptor(), getOutput2())); } } 
void MWFusedConvReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(QMgBqCuvjnbWHWiVPEwn)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(UpnEytIWGokwbTFkBcSx)); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(olKGEIcsxmLSoMhRhEtP)); if 
(vjDFlBZzKvbpPseAtMBP) { call_cuda_free(vjDFlBZzKvbpPseAtMBP); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(NMMfJylfQjiIUAKhXCJb)); if 
(MNuwXDSoGEYeABeVTwOh) { call_cuda_free(MNuwXDSoGEYeABeVTwOh); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(XLJXOFXdnZOyJvtltbyr != getLayer()->getInputTensor(0)) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(bUVPfnrJhLfHzOLUUrKk)); 
call_cuda_free(XLJXOFXdnZOyJvtltbyr->getData()); } if (getIsGrouped()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(cCXqPFPPcoHzYMDpnUxQ)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getGroupDescriptor())); } for(int idx 
= 0; idx < getLayer()->getNumOutputs(); idx++) { float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) call_cuda_free(data); 
} } return; } void MWFusedConvReLULayerImpl::loadWeights(const char* 
UKtMXCCqdjeyaVHabkxg) { MWFusedConvReLULayer* fusedConvReluLayer = 
static_cast<MWFusedConvReLULayer*>(getLayer()); FILE* UzaGmBLFEwmwaFXebUma = 
MWCNNLayer::openBinaryFile(UKtMXCCqdjeyaVHabkxg); assert(UzaGmBLFEwmwaFXebUma); 
assert(CGbFsczkgkhjcHoCKzBx == 
XLJXOFXdnZOyJvtltbyr->getChannels()/ClEhcJFlvGCgiavziIag); int fhikqqlnUKCjleVKDqiG = 
CGbFsczkgkhjcHoCKzBx*fusedConvReluLayer->getOutputTensor()->getChannels()*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs; 
 float* OKaRVOctKLlnIyGmjRNW = MALLOC_CALL(sizeof(float)*fhikqqlnUKCjleVKDqiG); 
fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), fhikqqlnUKCjleVKDqiG, UzaGmBLFEwmwaFXebUma); if( 
AzTsxYcYjIEJsGQbeYHm != 1 && BLjrjqvCcCommiXWQLjs != 1 ) { float* 
ONvcEjLBnVNUdjMKOAwF = 
MALLOC_CALL(sizeof(float)*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs); 
for(int k=0; k<fhikqqlnUKCjleVKDqiG/AzTsxYcYjIEJsGQbeYHm/BLjrjqvCcCommiXWQLjs; 
k++) { for(int i=0; i<AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs; i++) 
ONvcEjLBnVNUdjMKOAwF[i]=OKaRVOctKLlnIyGmjRNW[k*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs+i]; 
for(int j=0; j<AzTsxYcYjIEJsGQbeYHm; j++) for(int i=0; 
i<BLjrjqvCcCommiXWQLjs; i++) 
OKaRVOctKLlnIyGmjRNW[k*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs+j*BLjrjqvCcCommiXWQLjs+i]=ONvcEjLBnVNUdjMKOAwF[j+i*AzTsxYcYjIEJsGQbeYHm]; 
} free(ONvcEjLBnVNUdjMKOAwF); } CUDA_CALL(hipMemcpy(vjDFlBZzKvbpPseAtMBP, 
OKaRVOctKLlnIyGmjRNW, sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", UKtMXCCqdjeyaVHabkxg, fhikqqlnUKCjleVKDqiG, OKaRVOctKLlnIyGmjRNW[0]);
#endif
 free(OKaRVOctKLlnIyGmjRNW); fclose(UzaGmBLFEwmwaFXebUma); return; } void 
MWFusedConvReLULayerImpl::loadBias(const char* UKtMXCCqdjeyaVHabkxg) { 
MWFusedConvReLULayer* fusedConvReluLayer = 
static_cast<MWFusedConvReLULayer*>(getLayer()); FILE* UzaGmBLFEwmwaFXebUma = 
MWCNNLayer::openBinaryFile(UKtMXCCqdjeyaVHabkxg); assert(UzaGmBLFEwmwaFXebUma); int 
fhikqqlnUKCjleVKDqiG = fusedConvReluLayer->getOutputTensor()->getChannels();  float* 
OKaRVOctKLlnIyGmjRNW = MALLOC_CALL(sizeof(float)*fhikqqlnUKCjleVKDqiG); 
fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), fhikqqlnUKCjleVKDqiG, UzaGmBLFEwmwaFXebUma); 
CUDA_CALL(hipMemcpy(MNuwXDSoGEYeABeVTwOh, OKaRVOctKLlnIyGmjRNW, 
sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice)); 
free(OKaRVOctKLlnIyGmjRNW); fclose(UzaGmBLFEwmwaFXebUma); return; } void 
MWFusedConvReLULayerImpl::postSetup() { if(fYaOQTeunPwVjnhhTECh->getAutoTune()) 
{ getConvAlgoTuned(); } else if(!fYaOQTeunPwVjnhhTECh->getWorkSpace()) { 
getConvAlgoNoWorkSpace(); } hipdnnTensorDescriptor_t tmpInDesc = getIsGrouped() 
? cCXqPFPPcoHzYMDpnUxQ : bUVPfnrJhLfHzOLUUrKk; hipdnnTensorDescriptor_t 
juRPduBvIGpwaZiftkzr = getIsGrouped() ? *getGroupDescriptor() : 
*getOutputDescriptor(); size_t ugnnrhsgTeWucrMPCJUc; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
tmpInDesc, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, juRPduBvIGpwaZiftkzr, 
PmFfARVzoHVAYkfpuvqK, &ugnnrhsgTeWucrMPCJUc)); if( ugnnrhsgTeWucrMPCJUc > 
*fYaOQTeunPwVjnhhTECh->getPostSetupWorkSpaceSize() ) { 
fYaOQTeunPwVjnhhTECh->setPostSetupWorkSpaceSize(ugnnrhsgTeWucrMPCJUc); } } void 
MWFusedConvReLULayerImpl::getConvAlgoTuned() { hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t tempInDesc = getIsGrouped() ? 
cCXqPFPPcoHzYMDpnUxQ : bUVPfnrJhLfHzOLUUrKk; hipdnnTensorDescriptor_t 
juRPduBvIGpwaZiftkzr = getIsGrouped() ? *getGroupDescriptor() : 
*getOutputDescriptor(); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
tempInDesc, XLJXOFXdnZOyJvtltbyr->getData(), UpnEytIWGokwbTFkBcSx, vjDFlBZzKvbpPseAtMBP, 
QMgBqCuvjnbWHWiVPEwn, juRPduBvIGpwaZiftkzr, getData(), 3, &returnedAlgoCount, 
&perf_results[0], fYaOQTeunPwVjnhhTECh->getWorkSpace(), 
*fYaOQTeunPwVjnhhTECh->getWorkSpaceSize())); PmFfARVzoHVAYkfpuvqK = 
perf_results[0].algo; } void MWFusedConvReLULayerImpl::getConvAlgoNoWorkSpace() 
{ assert(fYaOQTeunPwVjnhhTECh->getWorkSpace() == 0); hipdnnTensorDescriptor_t 
tempInDesc = getIsGrouped() ? cCXqPFPPcoHzYMDpnUxQ : bUVPfnrJhLfHzOLUUrKk; 
hipdnnTensorDescriptor_t juRPduBvIGpwaZiftkzr = getIsGrouped() ? 
*getGroupDescriptor() : *getOutputDescriptor(); 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*fYaOQTeunPwVjnhhTECh->getCudnnHandle(), 
tempInDesc, UpnEytIWGokwbTFkBcSx, QMgBqCuvjnbWHWiVPEwn, juRPduBvIGpwaZiftkzr, 
HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, &PmFfARVzoHVAYkfpuvqK)); }